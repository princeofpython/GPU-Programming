#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<iostream>
#include <sys/time.h> 
#include<bits/stdc++.h>

using namespace std;

struct edgepairs{
  int x;
  int y;
};

bool compareTwoEdgePairs(edgepairs a, edgepairs b)
{
    if (a.x != b.x)
        return a.x < b.x;

    if (a.y != b.y)
        return a.y < b.y;
 
  return true;
}

// complete the following kernel...
__global__ void dkernel_Adds(int *gpuOA, int *gpuCA, int *gpulocals,int *gpucurrentupdate){
	unsigned node = blockIdx.x;
	unsigned vert = 0;
	vert = gpuOA[node];
	unsigned add = 0;
	for(; vert < gpuOA[node + 1]; ++vert ){
		add = add + gpucurrentupdate[gpuCA[vert]];
	}
	gpulocals[node] += add; 

}

// complete the following kernel...
__global__ void dkernel_Mins(int *gpuOA, int *gpuCA, int *gpulocals,int *gpucurrentupdate){
	unsigned node = blockIdx.x;
	unsigned vert = 0;

	vert = gpuOA[node];
	unsigned var = pow(2, 30);
	for(; vert < gpuOA[node + 1]; ++vert ){
		var = min(gpucurrentupdate[gpuCA[vert]], var);
	}
	gpulocals[node] = min(gpulocals[node], var);
}

// complete the following kernel...
__global__ void dkernel_Maxs(int *gpuOA, int *gpuCA, int *gpulocals,int *gpucurrentupdate){
	unsigned node = blockIdx.x;
	unsigned vert = 0;
	vert = gpuOA[node];
	unsigned var = 0;
	for(; vert < gpuOA[node + 1]; ++vert ){
		var = max(gpucurrentupdate[gpuCA[vert]], var);
	}
	gpulocals[node] = max(gpulocals[node], var);
}

int main(int argc,char **argv){

	//variable declarations
	int m,n;
	int number;
	int numofquery;
	int op;
	struct timeval t1, t2;
	vector <double> kerneltime;

	//File pointer declaration
	FILE *filePointer;

	//File Opening for read
	char *filename = argv[1]; 
    	filePointer = fopen( filename , "r") ; 
      
	//checking if file ptr is NULL
    	if ( filePointer == NULL ) 
    	{
        printf( "input.txt file failed to open." ) ; 
	      return 0;
    	}

	fscanf(filePointer, "%d", &n );		//scaning the number of vertices
        fscanf(filePointer, "%d", &m );		//scaning the number of edges

	//D.S to store the input graph in COO format
	vector <edgepairs> COO(m);
	
	//Reading from file and populate the COO
	for(int i=0 ; i<m ; i++ )
        {
		for(int j=0;j<2;j++){
			if ( fscanf(filePointer, "%d", &number) != 1)
            			break;
		if( j%2 == 0) 
		{       		
			if(number >= 1 && number <= 10000)
			COO[i].y = number;
		}		
		else
		{
			if(number >= 1 && number <= 10000)
			COO[i].x = number;
		}	

		}
        }
	// COO done...
	
	// sort the COO 
	sort(COO.begin(),COO.end(),compareTwoEdgePairs);
	//sorting COO done..
	
	// Converting the graph in COO format to CSR format..
	
	// create the CSR
	
	int *OA = (int *)malloc( (n+1)*sizeof(int));		//Offsets Array
	for(int i=0;i<n+1;i++){
                OA[i] = 0;
        }

	int *CA = (int *)malloc(m*sizeof(int));			//Coordinates Array
	OA[0]=0;

	//initialize the Coordinates Array
	for(int i=0;i<m;i++){
		if(COO[i].y >= 1 && COO[i].y <= 10000)
		CA[i] = COO[i].y - 1;
	}
	//initialize the Offsets Array
	for(int i=0;i<m;i++){
		if(COO[i].x >= 1 && COO[i].x <= 10000)
		OA[COO[i].x]++;		//store the frequency..
	}
	for(int i=0;i<n;i++){
		OA[i+1] += OA[i];	// do cumulative sum..
	}

	// Converting the graph to CSR done..
	
	// copy initial local values to the array from the file
	int *initlocalvals = (int *)malloc(n*sizeof(int));;
	for(int i=0 ; i<n ; i++ )
        {
        if ( fscanf(filePointer, "%d", &number) != 1)
            break;
         
        initlocalvals[i] = number;
        }
	// copying local vals end..

	// get number of queries from the file
	fscanf(filePointer, "%d", &numofquery);
	
	//copy OA,CA and initlocalvals to the GPU Memory
	int *gpuOA, *gpuCA, *gpulocals;
  hipMalloc( &gpuOA, sizeof(int) * (1+n) );
  hipMalloc( &gpuCA, sizeof(int) * m );
  hipMalloc( &gpulocals, sizeof(int) * n );
	hipMemcpy(gpuOA, OA, sizeof(int) * (1+n), hipMemcpyHostToDevice);
	hipMemcpy(gpuCA, CA, sizeof(int) * m, hipMemcpyHostToDevice);
	hipMemcpy(gpulocals, initlocalvals, sizeof(int) * n, hipMemcpyHostToDevice);


	int *currentupdate = (int *)malloc(n*sizeof(int));	// array to store the updates that are pushed by each vertex to there neighbors
	int *gpucurrentupdate;		// same as above but on GPU
  hipMalloc( &gpucurrentupdate, sizeof(int) * n );
  int *results = (int *)malloc(n*sizeof(int));         // storing the results from GPU to CPU for the enumerate query


  // open the output.txt to write the query results
      char *fname = argv[2]; 
      FILE *fptr;
      fptr = fopen(fname,"w");

	for(int i=0;i<numofquery;i++){

		//read the operator
		fscanf(filePointer, "%d", &op);

		if(op != 3){					// if operator is other then enumerate (i.e. +,min,max)

			// read the current updates in the array				
				for(int j=0 ; j<n ; j++ )
					{
						if ( fscanf(filePointer, "%d", &number) != 1)
							break;
						currentupdate[j] = number;
					}

			// copy current updates to gpu
			hipMemcpy(gpucurrentupdate, currentupdate, sizeof(int) * n, hipMemcpyHostToDevice);
			//kernel launches
		if(op == 0)	{
			gettimeofday(&t1, 0);	
			dkernel_Adds<<<n,1>>>(gpuOA,gpuCA,gpulocals,gpucurrentupdate);
			hipDeviceSynchronize();
			gettimeofday(&t2, 0);
			}
		if(op == 1)	{
			gettimeofday(&t1, 0);
			dkernel_Mins<<<n,1>>>(gpuOA,gpuCA,gpulocals,gpucurrentupdate);
			hipDeviceSynchronize();
			gettimeofday(&t2, 0);
			}
		if(op == 2)	{	
			gettimeofday(&t1, 0);
			dkernel_Maxs<<<n,1>>>(gpuOA,gpuCA,gpulocals,gpucurrentupdate);
			hipDeviceSynchronize();
			gettimeofday(&t2, 0);
			}
		
			double time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0; // Time taken by kernel in seconds 
			kerneltime.push_back(time);  

				printf("Time taken by kernel to execute is: %.6f ms\n", time); 
			}

		else{						// if operator is enumnerate then store the results to file
				//print local values of each vertices.
			hipMemcpy(results, gpulocals, n * sizeof(int), hipMemcpyDeviceToHost);  // get each locals from GPU
			for(int j=0;j<n;j++){
			fprintf(fptr ,"%d ", results[j] ); 
			}
			
			fprintf(fptr,"\n");
			/*
			for(int j=0;j<n + 1;j++){
				fprintf(fptr ,"%d ", OA[j] ); 
				}
			fprintf(fptr,"\n");
			for(int j=0;j<m;j++){
				fprintf(fptr ,"%d ", CA[j] ); 
				}
			fprintf(fptr,"\n"); 
			for(int j=0;j<m;j++){
				fprintf(fptr ,"%d %d ", COO[j].y, COO[j].x ); 
				}
			fprintf(fptr,"\n");
			*/
		}
		
	}

	int nall = kerneltime.size();
	double sumtime=0;
	for(int i=0;i<nall;i++){
		sumtime += kerneltime[i];
	}
	// print the time taken by all the kernels of the current test-case
	cout << "\ntotal time taken by the current test-case is " << sumtime << " ms\n";

  fclose(fptr);
  fclose(filePointer);

	return 0;
}
