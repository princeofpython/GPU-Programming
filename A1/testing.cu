#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1000

__global__ void per_row_kernel(int m,int n,int *A,int *B,int *C){  

    unsigned long long row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m){
        for(unsigned long long i = 0; i < n; ++i){
            C[row*n + i] = A[row*n + i] + B[row*n + i];  
        }
    }
}

__global__ void per_column_kernel(int m,int n,int *A,int *B,int *C){  
    unsigned long long col = (blockIdx.x * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    if (col < n){
        for(unsigned long long i = 0; i < m; ++i){
            C[i*n + col] = A[i*n + col] + B[i*n + col]; 
        }
    }
}  
__global__ void per_element_kernel(int m,int n,int *A,int *B,int *C){
    unsigned long long id = ((blockIdx.y*gridDim.x+blockIdx.x)*(blockDim.x*blockDim.y))+(threadIdx.y*blockDim.x+threadIdx.x);
    if (id < m*n){
        C[id] = A[id] + B[id];
    }
}

int main(){
    int A[N], B[N], C[N];
    for(int i = 0; i < N; ++i)
    {
       A[i] = i+1;
        B[i] = 2*i+2;
        C[i] = 0;
    }
    int* gpuA, *gpuB, *gpuC;
    hipMalloc(&gpuA, sizeof(int) * N);
    hipMalloc(&gpuB, sizeof(int) * N);
    hipMalloc(&gpuC, sizeof(int) * N);

    hipMemcpy(gpuA, A, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpuB, B, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpuC, C, sizeof(int) * N, hipMemcpyHostToDevice);

    per_element_kernel<<<10, 128>>>(20,50,gpuA,gpuB,gpuC);
    hipDeviceSynchronize();	
	hipMemcpy(C, gpuC, sizeof(int) * N, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; ++i)
    {
        printf("%d ",C[i]);
      if((i+1)%50==0)
        printf("\n");
    }

}