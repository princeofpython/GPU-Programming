
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <fstream>
#include <iomanip>
using namespace std;

bool checkOutput(string filename1, string filename2);
float studentKernelExecTime;

int main(int argc, char *argv[])
{
		if(argc != 3)
		{
				cout << "Usage: " << argv[0] << " <studentOutputFile> <seqOutputFile>" << endl;
				exit(0);
		}
		string studentOutputFile = argv[1];
		string seqOutputFile = argv[2];
		cout << fixed;
		cout << setprecision(6);
		bool isCorrect = checkOutput(studentOutputFile, seqOutputFile);
		if(isCorrect)
				cout << "Success " << endl;
		else
				cout << "Failure " << endl;
		return 0;
}

bool checkOutput(string studentOutputFile, string seqOutputFile)
{
	fstream studentFile(studentOutputFile.c_str(), ios_base::in);
	fstream baselineFile(seqOutputFile.c_str(), ios_base::in);
	int x, y;
	int flag=0;
	while(baselineFile >> x)
	{
		flag=1;
		studentFile >> y;
		if(x != y)
				return false;
	}
	if(flag==0)
	return false;

	return true;
}
