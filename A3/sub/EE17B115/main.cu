#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <algorithm>
using namespace std;

#define BLOCKSIZE 1024

__global__ void initialize(pair<float, int> * gputimes, unsigned n){
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n)
        gputimes[id].first = 0;
        gputimes[id].second = id;
}

__global__ void add_time(pair<float, int> * gputimes, unsigned vectorSize, int* gpuspeed, int i, int dis){
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < vectorSize)
        gputimes[id].first += 60* float(dis)/gpuspeed[i*vectorSize + gputimes[id].second];
}
__global__ void queue(pair<float, int> * gputimes, unsigned vectorSize, int x){
    unsigned id = threadIdx.x;
    for (int i =0; i < ceil(float(vectorSize)/blockDim.x); ++i){
        
        if(id + (i+1)* blockDim.x < vectorSize){
            if(gputimes[id + (i+1) * blockDim.x].first < gputimes[id + i * blockDim.x].first + x){
                gputimes[id + (i+1) * blockDim.x].first = gputimes[id + i * blockDim.x].first + x;
            }
        }
        gputimes[id + i * blockDim.x].first = gputimes[id + i * blockDim.x].first + x;
    }
}

//Complete the following function
void operations ( int n, int k, int m, int x, int dis, int *speed, int **results )  {
    pair<float, int> *times = (pair<float, int> *) malloc ( n * sizeof (pair<float, int>) );
    
    pair<float, int> *gputimes;
    int *gpuspeed;
    hipMalloc(&gputimes, n * sizeof (pair<float, int>));

    hipMalloc(&gpuspeed,  n*( k+1 ) * sizeof (int));
    hipMemcpy(gpuspeed, speed,  n*( k+1 ) * sizeof (int), hipMemcpyHostToDevice);

    unsigned nblocks = ceil(float(n) / BLOCKSIZE);
    //initialization
    hipMemcpy(gputimes, times, n * sizeof (pair<float, int>), hipMemcpyHostToDevice);
    initialize<<<nblocks, BLOCKSIZE>>>(gputimes, n);
    hipDeviceSynchronize();
    hipMemcpy(times, gputimes, n * sizeof (pair<float, int>), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < k+1; ++i){
        /*
        hipMemcpy(gputimes, times, n * sizeof (pair<float, int>), hipMemcpyHostToDevice);
        add_time<<<nblocks, BLOCKSIZE>>>(gputimes, n, gpuspeed, i, dis);
        hipMemcpy(times, gputimes, n * sizeof (pair<float, int>), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        */
        for(int j = 0; j< n; j++){
            times[j].first += 60* float(dis)/speed[i*n + times[j].second];
        }
        thrust::sort(thrust::host, times, times + n);
        //sort(times, times+ n);
        results[0][i] = times[0].second+1;
        results[1][i] = times[n-1].second+1;
        hipMemcpy(gputimes, times, n * sizeof (pair<float, int>), hipMemcpyHostToDevice);
        queue<<<1, m>>>(gputimes, n, x);
        hipDeviceSynchronize();
        hipMemcpy(times, gputimes, n * sizeof (pair<float, int>), hipMemcpyDeviceToHost);

    }

    for(int i = 0; i< n; ++i){
        results[2][times[i].second] = int(times[i].first) - x; 
    }

}

int main(int argc,char **argv){

    //variable declarations
    int n,k,m,x;
    int dis;
    
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0;
    }
    
    
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of vehicles
    fscanf( inputfilepointer, "%d", &k );      //scaning for number of toll tax zones
    fscanf( inputfilepointer, "%d", &m );      //scaning for number of toll tax points
    fscanf( inputfilepointer, "%d", &x );      //scaning for toll tax zone passing time
    
    fscanf( inputfilepointer, "%d", &dis );    //scaning for distance between two consecutive toll tax zones


    // scanning for speeds of each vehicles for every subsequent toll tax combinations
    int *speed = (int *) malloc ( n*( k+1 ) * sizeof (int) );
    for ( int i=0; i<=k; i++ )  {
        for ( int j=0; j<n; j++ )  {
            fscanf( inputfilepointer, "%d", &speed[i*n+j] );
        }
    }
    
    // results is in the format of first crossing vehicles list, last crossing vehicles list 
    //               and total time taken by each vehicles to pass the highway
    int **results = (int **) malloc ( 3 * sizeof (int *) );
    results[0] = (int *) malloc ( (k+1) * sizeof (int) );
    results[1] = (int *) malloc ( (k+1) * sizeof (int) );
    results[2] = (int *) malloc ( (n) * sizeof (int) );


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);


    // Function given to implement
    operations ( n, k, m, x, dis, speed, results );


    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    // First crossing vehicles list
    for ( int i=0; i<=k; i++ )  {
        fprintf( outputfilepointer, "%d ", results[0][i]);
    }
    fprintf( outputfilepointer, "\n");


    //Last crossing vehicles list
    for ( int i=0; i<=k; i++ )  {
        fprintf( outputfilepointer, "%d ", results[1][i]);
    }
    fprintf( outputfilepointer, "\n");


    //Total time taken by each vehicles to pass the highway
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", results[2][i]);
    }
    fprintf( outputfilepointer, "\n");

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    return 0;
}